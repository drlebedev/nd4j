extern "C"

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

__global__ void div_double(int n, double *a, double *b, double *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
      sum[i] = a[i] / b[i];


}